#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <ctime>

struct HashGenerator {
  int current_;
  HashGenerator (int start) : current_(start) {}
  double operator() () { current_++; 
                      return generateReal(current_);}
  // generates a int hash from an int value 
  int hash(int i)
  {
    uint64_t v = ((uint64_t) i) * 3935559000370003845 + 2691343689449507681;
    v = v ^ (v >> 21);
    v = v ^ (v << 37);
    v = v ^ (v >> 4);
    v = v * 4768777513237032717;
    v = v ^ (v << 20);
    v = v ^ (v >> 41);
    v = v ^ (v <<  5);
    return (int) (v & ((((uint64_t) 1) << 31) - 1));
  }

  // generates a pseudorandom double precision real from an integer
  double generateReal(int i) {
      return (double(hash(i)));
  }
};

struct IncGenerator {
    double current_;
    IncGenerator (double start) : current_(start) {}
    double operator() () { return current_++; }
};

void parallel_sort(){
  
}

int main(void)
{
  int N = 10000000;
  // generate 100M numbers serially
  thrust::host_vector<double> h_vec(N);
  thrust::host_vector<double> v_vec(N);
  thrust::host_vector<double> h_vec_result(N);
  thrust::host_vector<double> v_vec_result(N);
  HashGenerator HG (0);
  IncGenerator IG (0);

  clock_t begin_generation = clock();
  std::generate(h_vec.begin(), h_vec.end(), HG);
  std::generate(v_vec.begin(), v_vec.end(), IG);
  clock_t end_generation = clock();
  double generation_time = double(end_generation - begin_generation) / CLOCKS_PER_SEC;
  std::cout << "Generation Time: " << generation_time << std::endl;

  int numRuns = 5;

  for(int i = 0; i < numRuns; i++) {
    
    clock_t begin_sort_copy = clock();
    // transfer data to the device
    thrust::device_vector<double> d_vec = h_vec;
    thrust::device_vector<double> dv_vec = v_vec;

    clock_t begin_sort = clock();
    // sort data on the device
    prallel_sort(thrust::device, d_vec.begin(), d_vec.end(), dv_vec.begin());
    hipDeviceSynchronize();
    clock_t end_sort = clock();

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec_result.begin());
    thrust::copy(dv_vec.begin(), dv_vec.end(), v_vec_result.begin());
    hipDeviceSynchronize();
    clock_t end_sort_copy = clock();

    double sort_copy_time = double(end_sort_copy - begin_sort_copy) / CLOCKS_PER_SEC;
    double sort_time = double(end_sort - begin_sort) / CLOCKS_PER_SEC;
    std::cout << "Sort + Copy Time: " << sort_copy_time << std::endl;
    std::cout << "Sort Only Time: " << sort_time << std::endl;

    clock_t begin_check = clock();
    for(int j = 1; j < N; j++) {
      if(h_vec_result[j] < h_vec_result[j-1]){
        std::cout << "Error: " << h_vec_result[j-1] << " is before " << h_vec_result[j] << std::endl;
      }
    }
    clock_t end_check = clock();
    double check_time = double(end_check - begin_check) / CLOCKS_PER_SEC;
    std::cout << "Check Time: " << check_time << std::endl;
  }

  return 0;
}